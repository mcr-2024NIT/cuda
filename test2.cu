#include "hip/hip_runtime.h"
    #include <iostream>
    #include <Cudacpp/cudaVector.cuh>
    #include "lib/atomicQ.cuh"
    #include "lib/atomicMap.cuh"
    #include <hip/hip_runtime.h>
    #include <chrono>

    const int WIDTH = 10;
    const int HEIGHT = 20;


    struct TetrisState {
        cudacpp::cudaVector<cudacpp::cudaVector<int>> board;
        int x, y, rotation;
        cudacpp::cudaVector<char> moves;

        __device__ bool operator==(const TetrisState& other) const {
            return board == other.board && x == other.x && y == other.y && rotation == other.rotation;
        }
    };


    struct TetrisStateHash {
        __device__ size_t operator()(const TetrisState& state) const {
            size_t seed = 0;
            for (int i = 0; i < state.board.size(); ++i) {
                for (int j = 0; j < state.board[i].size(); ++j) {
                    seed ^= state.board[i][j] + 0x9e3779b9 + (seed << 6) + (seed >> 2);
                }
            }
            seed ^= state.x + 0x9e3779b9 + (seed << 6) + (seed >> 2);
            seed ^= state.y + 0x9e3779b9 + (seed << 6) + (seed >> 2);
            seed ^= state.rotation + 0x9e3779b9 + (seed << 6) + (seed >> 2);
            return seed;
        }
    };

    __device__ cudacpp::cudaVector<cudacpp::cudaVector<int>> rotatePiece(const cudacpp::cudaVector<cudacpp::cudaVector<int>>& piece, int rotation) {
        int n = piece.size();
        cudacpp::cudaVector<cudacpp::cudaVector<int>> rotatedPiece(n, cudacpp::cudaVector<int>(n, 0));
        for (int i = 0; i < n; ++i) {
            for (int j = 0; j < n; ++j) {
                if (rotation == 1) rotatedPiece[j][n - 1 - i] = piece[i][j];
                else if (rotation == 2) rotatedPiece[n - 1 - i][n - 1 - j] = piece[i][j];
                else if (rotation == 3) rotatedPiece[n - 1 - j][i] = piece[i][j];
                else rotatedPiece[i][j] = piece[i][j];
            }
        }
        return rotatedPiece;
    }

    __device__
    bool isValid(const TetrisState& state, const cudacpp::cudaVector<cudacpp::cudaVector<int>>& piece) {
        auto rotatedPiece = rotatePiece(piece, state.rotation);
        for (int i = 0; i < rotatedPiece.size(); ++i) {
            for (int j = 0; j < rotatedPiece[0].size(); ++j) {
                if (rotatedPiece[i][j]) {
                    int newX = state.x + j;
                    int newY = state.y + i;
                    if (newX < 0 || newX >= WIDTH || newY < 0 || newY >= HEIGHT || state.board[newY][newX]) {
                        return false;
                    }
                }
            }
        }
        return true;
    }

    __device__
    cudacpp::cudaVector<cudacpp::cudaVector<int>> placePiece(cudacpp::cudaVector<cudacpp::cudaVector<int>> board, const cudacpp::cudaVector<cudacpp::cudaVector<int>>& piece, int x, int y, int rotation) {
        auto rotatedPiece = rotatePiece(piece, rotation);
        for (int i = 0; i < rotatedPiece.size(); ++i) {
            for (int j = 0; j < rotatedPiece[0].size(); ++j) {
                if (rotatedPiece[i][j] && y + i < HEIGHT && x + j < WIDTH) {
                    board[y + i][x + j] = rotatedPiece[i][j];
                }
            }
        }
        return board;
    }
    __device__
    void printBoard(cudacpp::cudaVector<cudacpp::cudaVector<int>> board) {
        for (int i = 0; i < board.size(); ++i) {
            for (int j = 0; j < board[i].size(); ++j) {
                printf("%d ", board[i][j]);
            }
            printf("\n");
        }
        printf("\n");
    }
    //各行、上から確認して一番初めに要素が確認されたところでストップしてその行数を出力する
    __device__
    void isTopRowAllZero(TetrisState& state) {
        for (int i = 0; i < state.board.size(); ++i) {
            for (int j = 0; j < state.board[i].size(); ++j) {
                if (state.board[i][j]) {
                    state.y = i - 1;
                    printf("y: %d\n", state.y);
                    return ;
                }
            }
        }
        state.y = state.board.size()-1;
        return ;
    }



    __global__
    void bfsAllTetrisStates(int* devicePiece, int* deviceBoard) {
        int tid = threadIdx.x;
        
            printf("tid: %d\n", tid);        
        extern __shared__ int sharedMem[];

    cudacpp::cudaQueue<TetrisState, 196>* q = (cudacpp::cudaQueue<TetrisState, 196>*)sharedMem;
    size_t queueSize = sizeof(cudacpp::cudaQueue<TetrisState, 196>);
    printf("queueSize: %d\n", queueSize);
    cudacpp::cuda_unordered_map<TetrisState, cudacpp::cudaVector<char>, TetrisStateHash>* uniqueBoards =
        new (&sharedMem[queueSize]) cudacpp::cuda_unordered_map<TetrisState, cudacpp::cudaVector<char>, TetrisStateHash>(256);
        printf("uniqueBoards size: %d\n", 6);
    size_t mapSize = sizeof(cudacpp::cuda_unordered_map<TetrisState, cudacpp::cudaVector<char>, TetrisStateHash>);
    printf("mapSize: %d\n", mapSize);
    cudacpp::cudaVector<TetrisState>* finalStates =
        new (&sharedMem[queueSize + mapSize]) cudacpp::cudaVector<TetrisState>();
        cudacpp::cudaVector<cudacpp::cudaVector<int>> initialBoard(HEIGHT, cudacpp::cudaVector<int>(WIDTH, 0));
        cudacpp::cudaVector<cudacpp::cudaVector<int>> piece(3, cudacpp::cudaVector<int>(3, 0));
        if (tid==0)
        {
            for (int i = 0; i < HEIGHT; ++i) {
                for (int j = 0; j < WIDTH; ++j) {
                    initialBoard[i][j] = deviceBoard[i * WIDTH + j];
                }
            }

            for (int i = 0; i < 3; ++i) {
                for (int j = 0; j < 3; ++j) {
                    piece[i][j] = devicePiece[i * 3 + j];
                }
            }
            TetrisState initialState = {initialBoard, WIDTH / 2-2, 0, 0, {}};

            isTopRowAllZero(initialState);
            initialState.y-=piece.size();
        
            q->push(initialState);
        }
        __syncthreads();


        






        int is = 0;
        while (tid < q->size()) {

                    printf("q size: %d\n", q->size());
            printf("tid: %d\n", tid);
            
            
            TetrisState currentState;
            q->pop_and_front(currentState);
            if (currentState.y == HEIGHT - 1 || !isValid({currentState.board, currentState.x, currentState.y + 1, currentState.rotation, currentState.moves}, piece)) {
                currentState.board = placePiece(currentState.board, piece, currentState.x, currentState.y, currentState.rotation);
                currentState.moves.push_back('P');
                auto it = uniqueBoards->find(currentState);
                if (it== uniqueBoards->end()||it->second.size() > currentState.moves.size()) {
                    (*uniqueBoards)[currentState] = currentState.moves;
                    finalStates->push_back(currentState);
                    for (int i = 0; i < currentState.moves.size(); i++)
                    {
                        printf("%c ", currentState.moves[i]);
                    }
                    
                }
                
                continue;
            }
            auto it = uniqueBoards->find(currentState);
            printf("uniqueBoards size: %d\n", 6);
            if (it== uniqueBoards->end()||it->second.size() > currentState.moves.size()) {

               (*uniqueBoards)[currentState] = currentState.moves;


                TetrisState newState = currentState;
                newState.rotation = (currentState.rotation + 1) % 4;
                newState.moves.push_back('O');
                if (isValid(newState, piece)) {
                    q->push(newState);
                }
                if (currentState.moves.back() != 'R'){

                
                    TetrisState newStateL = currentState;
                    newStateL.x--;
                    newStateL.moves.push_back('L');
                    if (isValid(newStateL, piece)) {
                        q->push(newStateL);
                        
                    }
                }
                if (currentState.moves.back() != 'L'){
                    TetrisState newStateR = currentState;
                    newStateR.x++;
                    newStateR.moves.push_back('R');
                    if (isValid(newStateR, piece)) {
                        q->push(newStateR);
                    }

                }

                TetrisState newStateD = currentState;
                newStateD.y++;
                newStateD.moves.push_back('D');
                if (isValid(newStateD, piece)) {


                    q->push(newStateD);
                }
                printf("q size: %d\n", q->size());
            
            }
            __syncthreads();
            


            
        is++;
        }

        for (const TetrisState& state : *finalStates) {
            //printBoard(state.board);
            
            for (const auto& move : state.moves) {
                printf("%c ", move);
            }
            printf("\n");
        }
        printf("Number of unique states: %d\n", finalStates->size());

    }

    int main() {
        cudacpp::cudaVector<cudacpp::cudaVector<int>> initialBoard(HEIGHT, cudacpp::cudaVector<int>(WIDTH, 0));
    cudacpp::cudaVector<cudacpp::cudaVector<int>> piece = {
            {0, 1, 0},
            {1, 1, 1},
            {0, 0, 0}
        };

        // ホスト側のデータを定義し、データを準備
        cudacpp::cudaVector<int> hostBoard(HEIGHT*WIDTH);
        cudacpp::cudaVector<int> hostPiece(piece.size()*piece[0].size());
        for (int i = 0; i < HEIGHT; ++i) {
            for (int j = 0; j < WIDTH; ++j) {
                hostBoard[i * WIDTH + j] = initialBoard[i][j];
            }
        }

        for (int i = 0; i < piece.size(); ++i) {
            for (int j = 0; j < piece[0].size(); ++j) {
                hostPiece[i * piece[0].size() + j] = piece[i][j];
            }
        }



        // デバイス側のメモリ確保
        int* devicePiece;int* deviceBoard;
        hipMalloc(&devicePiece, piece.size()*piece[0].size() * sizeof(int));
        hipMalloc(&deviceBoard, HEIGHT*WIDTH * sizeof(int));

        // ホストからデバイスへのデータ転送
        hipMemcpyAsync(devicePiece, hostPiece.data(), piece.size() * piece[0].size() * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpyAsync(deviceBoard, hostBoard.data(), HEIGHT * WIDTH * sizeof(int), hipMemcpyHostToDevice);

        // カーネルの設定（スレッド1つ、ブロック1つ）
        auto start = std::chrono::system_clock::now();
        int size = sizeof(cudacpp::cudaQueue<TetrisState,196>) + sizeof(cudacpp::cuda_unordered_map<TetrisState, cudacpp::cudaVector<char>, TetrisStateHash>)+sizeof(cudacpp::cudaVector<TetrisState>);

        bfsAllTetrisStates<<<1, 1,size>>>(devicePiece, deviceBoard);
            // カーネルの実行を待機
        hipDeviceSynchronize();
        auto end = std::chrono::system_clock::now();
        auto dur = end - start;
        auto msec = std::chrono::duration_cast<std::chrono::milliseconds>(dur).count();
        std::cout << msec << " ms" << std::endl;



        // デバイスメモリの解放
        hipFree(devicePiece);
        hipFree(deviceBoard);


        return 0;
    }
